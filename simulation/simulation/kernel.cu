#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/device_functions.h>
#include <hip/hip_runtime_api.h>

const int NX = 10;      // X size
const int NY = 10;      // y size

const int MAX_ITER = 1000;  

__global__ void Laplace(float *T_old, float *T_new)
{
	float klamda = 0.1;

	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;

	 
	int point = i + j*NX;           
	int top = i + (j + 1)*NX;       
	int down = i + (j - 1)*NX;       
	int right = (i + 1) + j*NX;       
	int left = (i - 1) + j*NX;       
								  
	if (i>0 && i<NX-1  && j>0 && j<NY-1) {
		T_new[point] = klamda*(T_old[right] + T_old[left] + T_old[top] + T_old[down]);
	}
	
		
	
}

// initialization

void Initialize(float *TEMPERATURE)
{
	for (int i = 0; i<NX; i++) {
		for (int j = 0; j<NY; j++) {
			int index = i + j*NX;
			TEMPERATURE[index] = 0.0;
		}
	}

	// set boundary condition

	for (int j = 0; j<NY; j++) {
		int index = j*NX;
		TEMPERATURE[index] = 1000;
	}
	for (int j = 0; j<NY; j++) {
		int index = j;
		TEMPERATURE[index] = 2000;
	}
	//9,14,19
	for (int j = (NY*2)-1; j<NY*NX; j+=NY) {
		int index = j;
		TEMPERATURE[index] = 1000;
	}
	for (int j = (NY *(NX-1) ) ; j<NY*NX; j++) {
		int index = j;
		TEMPERATURE[index] = 1000;
	}
}

int main(int argc, char **argv)
{
	        
	float *_T1, *_T2;  
					   
	float *T = new float[NX*NY];
	std::cout << " start \n";
	Initialize(T);
	for (int j = NY - 1; j >= 0; j--) {
		for (int i = 0; i<NX; i++) {
			int index = i + j*NX;
			std::cout << T[index] << " ";
		}
		std::cout << std::endl;
	}


	hipMalloc((void **)&_T1, NX*NY * sizeof(float));
	hipMalloc((void **)&_T2, NX*NY * sizeof(float));

	hipMemcpy(_T1, T, NX*NY * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(_T2, T, NX*NY * sizeof(float), hipMemcpyHostToDevice);

	int ThreadsPerBlock = 16;
	dim3 dimBlock(ThreadsPerBlock, ThreadsPerBlock);
	dim3 dimGrid(ceil(float(NX) / float(dimBlock.x)), ceil(float(NY) / float(dimBlock.y)), 1);

	for (size_t i = 0; i < MAX_ITER/2; i++)
	{
		Laplace << <dimGrid, dimBlock >> >(_T1, _T2);
		Laplace << <dimGrid, dimBlock >> >(_T2, _T1);
	}
		
	
	hipMemcpy(T, _T2, NX*NY * sizeof(float), hipMemcpyDeviceToHost);
	hipDeviceSynchronize();
	std::cout << " result \n";
	// print the results to screen
	for (int j=NY-1;j>=0;j--) {
	for (int i=0;i<NX;i++) {
	int index = i + j*NX;
	std::cout << T[index] << " ";
	}
	std::cout << std::endl;
	}
	
	delete T;

	hipFree(_T1);
	hipFree(_T2);

	return 0;
}
